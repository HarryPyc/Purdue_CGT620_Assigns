
#include "hip/hip_runtime.h"


#include <stdio.h>

#define EXPLICIT_MEMORY_CONTROL

const int n = 1e5;
void FillArray(int *a, int *b) {
	for (int i = 0; i < n; i++)
		a[i] = i, b[i] = i*10;
}
__global__ void ArrayAdd(const int* d_a, const int* d_b, int* d_c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		d_c[i] = d_a[i] + d_b[i];
		//printf("a[%i] = %i, b[%i] = %i, c[%i] = %i\n", i, d_a[i], i, d_b[i], i, d_c[i]);
	}
}

int main() {
	int *h_a = new int[n], *h_b = new int[n], *h_c = new int[n];
	int* d_a, * d_b, * d_c;
	hipEvent_t startT, stopT;
	float time;
	hipEventCreate(&startT);
	hipEventCreate(&stopT);
	hipEventRecord(startT, 0);
#ifdef EXPLICIT_MEMORY_CONTROL
	FillArray(h_a, h_b);
	hipMalloc((void**)&d_a, n * sizeof(int));
	hipMalloc((void**)&d_b, n * sizeof(int));
	hipMalloc((void**)&d_c, n * sizeof(int));
	hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n * sizeof(int), hipMemcpyHostToDevice);
#else
	cudaMallocManaged(&d_a, n * sizeof(int));
	cudaMallocManaged(&d_b, n * sizeof(int));
	cudaMallocManaged(&d_c, n * sizeof(int));
	FillArray(d_a, d_b);
#endif
	ArrayAdd <<<n/64+1, 64>>> (d_a, d_b, d_c);
	if (hipGetLastError() != hipSuccess)
		printf("Launch Kernal Failed\n");

	hipEventRecord(stopT, 0);
	hipEventSynchronize(stopT);
	hipEventElapsedTime(&time, startT, stopT);
	hipEventDestroy(startT);
	hipEventDestroy(stopT);
	printf("time used: %f miliseconds", time);
	delete[] h_a, h_b, h_c;
	hipFree(d_c);
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}